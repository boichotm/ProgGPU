#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"

#define THREADS 4096
#define TAB_SIZE 8192

__global__ void kernel(int *a, int *b, int *c) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < TAB_SIZE) c[tid] = a[tid] + b[tid];
}

int main(int argc, char **argv)
{
    int sz_in_bytes = sizeof(int) * TAB_SIZE;

    int *h_c;
    int res = 0;
    int *d_a, *d_b, *d_c;

    // Allocation on host (malloc)
    h_c = (int *)malloc(sz_in_bytes);

    // Allocation on device (hipMalloc)
    checkCudaErrors(hipMalloc((void **)&d_a, sz_in_bytes));
    checkCudaErrors(hipMalloc((void **)&d_b, sz_in_bytes));
    checkCudaErrors(hipMalloc((void **)&d_c, sz_in_bytes));

    checkCudaErrors(hipMemset(d_a, 1, sz_in_bytes));
    checkCudaErrors(hipMemset(d_b, 2, sz_in_bytes));

    // Kernel configuration
    dim3 dimBlock(THREADS, 1, 1);
    dim3 dimGrid(TAB_SIZE / THREADS + 1, 1, 1);

    // Kernel launch
    kernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);

    // Retrieving data from device (hipMemcpy)
    checkCudaErrors(hipMemcpy(h_c, d_c, sz_in_bytes, hipMemcpyDeviceToHost));

    // Freeing on device (hipFree)
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));

    // computing sum of tab element
    for (int i = 0; i < TAB_SIZE; i++) res += h_c[i];

    // Verifying if
    if (res == 3 * TAB_SIZE) {
        fprintf(stderr, "TEST PASSED !\n");
    }
    else
    {
        fprintf(stderr, "TEST FAILED !\n");
    }

    free(h_c);

    return 0;
}
