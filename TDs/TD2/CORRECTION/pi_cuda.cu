#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define gettime(t) clock_gettime(CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9*(double)t.tv_nsec)
/** return time in second
*/
double get_elapsedtime(void)
{
  struct timespec st;
  int err = gettime(&st);
  if (err !=0) return 0;
  return (double)st.tv_sec + get_sub_seconde(st);
}

/* QUESTION 3 */
#define TRIALS_PER_THREAD 4096
#define BLOCKS 512
#define THREADS 256
/* FIN QUESTION 3*/

/* QUESTION 6 */
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int points_in_circle = 0;
  float x = 0., y = 0.;

  hiprand_init(2020, tid, 0, &states[tid]);  //   Initialize CURAND

  for(int i = 0; i < TRIALS_PER_THREAD; i++)
  {
    x = hiprand_uniform (&states[tid]);
    y = hiprand_uniform (&states[tid]);
    points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
  }
  estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}
/* FIN QUESTION 6 */

int main (int argc, char *argv[]) {
  float h_counts[BLOCKS * THREADS] = { 0 };
  double t0 = 0., t1 = 0., duration = 0.;

  printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD, BLOCKS, THREADS);

/* QUESTION 4 */
  float *d_counts;
  hipMalloc((void **) &d_counts, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts
/* FIN QUESTION 4 */

/* QUESTION 6 */
  hiprandState *d_states;
  hipMalloc( (void **)&d_states, THREADS * BLOCKS * sizeof(hiprandState) );
/* FIN QUESTION 6 */

  t0 = get_elapsedtime();
/* QUESTION 3 */
  gpu_monte_carlo<<<BLOCKS, THREADS>>>(d_counts, d_states);
/* FIN QUESTION 3*/

/* QUESTION 5 */
  hipMemcpy(h_counts, d_counts, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results 
/* FIN QUESTION 5 */

  float pi_gpu = 0.f;
  for(int i = 0; i < BLOCKS * THREADS; i++)
  {
    pi_gpu += h_counts[i];
  }

  pi_gpu /= (BLOCKS * THREADS);

  t1 = get_elapsedtime();
  duration = (t1 - t0);

  printf("GPU pi calculated in %lf s.\n", duration);
  fprintf(stdout, "Pi ~= %lf\n", pi_gpu);

  return 0;
}
